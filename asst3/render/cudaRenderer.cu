#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"
#include "CycleTimer.h"

#define NUM_PIECE_WIDTH 32
#define NUM_PIECE_HEIGHT 32
#define THREADS_PER_BLOCK 256

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;

    // 分块
    int* numElementInPiece;
    int* numElementInPieceCurrent;
    int* prefixSum;
    int numPieceWidth;
    int numPieceHeight;
    int widthPerPiece;
    int heightPerPiece;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(index, pixelCenterNorm, p, imgPtr);
            imgPtr++;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
    cudaDeviceElementInPiece = NULL;
    cudaDevicePrefixSum = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
        hipFree(cudaDeviceElementInPiece);
        hipFree(cudaDeviceElementInPieceCurrent);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    // 具体分块信息
    numPieceWidth = NUM_PIECE_WIDTH;
    numPieceHeight = NUM_PIECE_HEIGHT;
    widthPerPiece = (image->width + numPieceWidth - 1) / numPieceWidth;
    heightPerPiece = (image->height + numPieceHeight - 1) / numPieceHeight;
    printf("widthPerPiece: %d, heightPerPiece: %d\n", widthPerPiece, heightPerPiece);
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&cudaDeviceElementInPiece, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");
    cudaStatus = hipMalloc((void**)&cudaDeviceElementInPieceCurrent, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");
    cudaStatus = hipMalloc((void**)&cudaDevicePrefixSum, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");

    hipMemset(cudaDeviceElementInPiece, 0, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMemset failed\n");
    hipMemset(cudaDeviceElementInPieceCurrent, 0, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMemset failed\n");

    hipDeviceSynchronize();

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    params.numPieceWidth = numPieceWidth;
    params.numPieceHeight = numPieceHeight;
    params.widthPerPiece = widthPerPiece;
    params.heightPerPiece = heightPerPiece;
    params.numElementInPiece = cudaDeviceElementInPiece;
    params.numElementInPieceCurrent = cudaDeviceElementInPieceCurrent;
    // params.newMemoryToPiece = cudaDeviceNewMemoryToPiece;
    params.prefixSum = cudaDevicePrefixSum;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();                                                                     
}

__device__ __inline__ int
circleInBox(
    float circleX, float circleY, float circleRadius,
    float boxL, float boxR, float boxT, float boxB)
{

    // clamp circle center to box (finds the closest point on the box)
    float closestX = (circleX > boxL) ? ((circleX < boxR) ? circleX : boxR) : boxL;
    float closestY = (circleY > boxB) ? ((circleY < boxT) ? circleY : boxT) : boxB;

    // is circle radius less than the distance to the closest point on
    // the box?
    float distX = closestX - circleX;
    float distY = closestY - circleY;

    if ( ((distX*distX) + (distY*distY)) <= (circleRadius*circleRadius) ) {
        return 1;
    } else {
        return 0;
    }
}

__global__
void numCircleInPiece() {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= cuConstRendererParams.numCircles) {
        return;
    }
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short numPieceWidth = cuConstRendererParams.numPieceWidth;
    short numPieceHeight = cuConstRendererParams.numPieceHeight;
    short widthEachPiece = (imageWidth + numPieceWidth - 1) / numPieceWidth;
    short heightEachPiece = (imageHeight + numPieceHeight - 1) / numPieceHeight;
    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index * 3]);
    float  rad = cuConstRendererParams.radius[index];

    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    short pieceMinX = screenMinX / widthEachPiece;
    short pieceMaxX = (screenMaxX - 1) / widthEachPiece + 1;
    short pieceMinY = screenMinY / heightEachPiece;
    short pieceMaxY = (screenMaxY - 1) / heightEachPiece + 1;

    float invWidth = 1.f / imageWidth * widthEachPiece;
    float invHeight = 1.f / imageHeight * heightEachPiece;

    // 遍历像素块
    float boxB = invHeight * pieceMinY;
    for (int pieceY = pieceMinY; pieceY < pieceMaxY; pieceY++, boxB += invHeight) {
        int pieceIndex = pieceY * numPieceWidth + pieceMinX;
        int* val = &cuConstRendererParams.numElementInPiece[pieceIndex];
        float boxL = invWidth * pieceMinX;
        for (int pieceX = pieceMinX; pieceX < pieceMaxX; pieceX++, val++, boxL += invWidth) {
            if (circleInBox(p.x, p.y, rad, boxL, boxL + invWidth, boxB + invHeight, boxB)) {
                atomicAdd(val, 1);
            }
        }
    }
}

__global__
void scan_kernel_upsweep(int* output, int two_dplus1, int N) {
    long long index = threadIdx.x + blockIdx.x * blockDim.x;
    index *= two_dplus1;
    if (index < N) {
        int two_d = two_dplus1 >> 1;
        output[index + two_dplus1 - 1] += output[index + two_d - 1];
    }
}

__global__
void scan_kernel_downsweep(int* output, int two_dplus1, int N) {
    long long index = threadIdx.x + blockIdx.x * blockDim.x;
    index *= two_dplus1;
    if (index < N) {
        int two_d = two_dplus1 >> 1;
        int t = output[index + two_d - 1];
        output[index + two_d - 1] = output[index + two_dplus1 - 1];
        output[index + two_dplus1 - 1] += t;
    }
}
// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}
int exclusive_scan(int* input, int N, int* result)
{
    assert(N == nextPow2(N));
    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocks;
    int res = 0;
    hipMemcpy(&res, result + N - 1, sizeof(int), hipMemcpyDeviceToHost);
    
    // 实现 1
    for (int two_d = 1, num = (N >> 1); two_d <= (N >> 1); two_d <<= 1, num >>= 1) {
        blocks = (num + threadsPerBlock - 1) / threadsPerBlock;
        scan_kernel_upsweep<<<blocks, threadsPerBlock>>>(result, two_d << 1, N);
    }
    
    int t = 0;
    hipMemcpy(result + N - 1, &t, sizeof(int), hipMemcpyHostToDevice);

    for (int two_d = (N >> 1), num = 1; two_d >= 1; two_d >>= 1, num <<= 1) {
        blocks = (num + threadsPerBlock - 1) / threadsPerBlock;
        scan_kernel_downsweep<<<blocks, threadsPerBlock>>>(result, two_d << 1, N);
    }
    hipDeviceSynchronize();

    hipMemcpy(&t, result + N - 1, sizeof(int), hipMemcpyDeviceToHost);
    return res + t;
}

__global__
void addIndexToNewMemory(int *allIndexs) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= cuConstRendererParams.numCircles) {
        return;
    }
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short numPieceWidth = cuConstRendererParams.numPieceWidth;
    short numPieceHeight = cuConstRendererParams.numPieceHeight;
    short widthEachPiece = (imageWidth + numPieceWidth - 1) / numPieceWidth;
    short heightEachPiece = (imageHeight + numPieceHeight - 1) / numPieceHeight;
    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index * 3]);
    float  rad = cuConstRendererParams.radius[index];

    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    short pieceMinX = screenMinX / widthEachPiece;
    short pieceMaxX = (screenMaxX - 1) / widthEachPiece + 1;
    short pieceMinY = screenMinY / heightEachPiece;
    short pieceMaxY = (screenMaxY - 1) / heightEachPiece + 1;

    float invWidth = 1.f / imageWidth * widthEachPiece;
    float invHeight = 1.f / imageHeight * heightEachPiece;
    // 新的地址空间合集

    float boxB = invHeight * pieceMinY;
    // 遍历像素块
    for (int pieceY = pieceMinY; pieceY < pieceMaxY; pieceY++, boxB += invHeight) {
        int pieceIndex = pieceY * numPieceWidth + pieceMinX;
        float boxL = invWidth * pieceMinX;
        for (int pieceX = pieceMinX; pieceX < pieceMaxX; pieceX++, pieceIndex++, boxL += invWidth) {
            if (circleInBox(p.x, p.y, rad, boxL, boxL + invWidth, boxB + invHeight, boxB)) {
                int* val = &cuConstRendererParams.numElementInPieceCurrent[pieceIndex];
                int priorNum = cuConstRendererParams.prefixSum[pieceIndex];
                // 修改部分
                int d = atomicAdd(val, 1);
                allIndexs[priorNum + d] = index;
            }
        }
    }
}
__global__
void sortPerPiece(int *allIndexs) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int numPieceWidth = cuConstRendererParams.numPieceWidth;
    int numPieceHeight = cuConstRendererParams.numPieceHeight;
    if (index >= numPieceWidth * numPieceHeight) {
        return;
    }
    // 获取地址
    int total = cuConstRendererParams.numElementInPieceCurrent[index];
    int priorNum = cuConstRendererParams.prefixSum[index];
    allIndexs += priorNum;
    // 特判，会出现一个地方的空间为 0 的情况;
    if (total <= 0) {
        return;
    }
    // 堆排序
    // 变大根堆
    for (int i = total - 1;i >= 0;i--) {
        int root = i;
        while(true) {
            int sonL = (root << 1) + 1;
            int sonR = sonL + 1;
            if (sonL >= total) {
                break;
            }
            if (sonR >= total || allIndexs[sonL] >= allIndexs[sonR]) {
                if (allIndexs[root] < allIndexs[sonL]) {
                    sonR = allIndexs[sonL];
                    allIndexs[sonL] = allIndexs[root];
                    allIndexs[root] = sonR;
                    root = sonL;
                } else {
                    break;
                }
            } else {
                if (allIndexs[root] < allIndexs[sonR]) {
                    sonL = allIndexs[sonR];
                    allIndexs[sonR] = allIndexs[root];
                    allIndexs[root] = sonL;
                    root = sonR;
                } else {
                    break;
                }
            }
        }
    }

    while(--total) {
        int root = allIndexs[0];
        allIndexs[0] = allIndexs[total];
        allIndexs[total] = root;
        root = 0;
        while(true) {
            int sonL = (root << 1) + 1;
            int sonR = sonL + 1;
            if (sonL >= total) {
                break;
            }
            if (sonR >= total || allIndexs[sonL] >= allIndexs[sonR]) {
                if (allIndexs[root] < allIndexs[sonL]) {
                    sonR = allIndexs[sonL];
                    allIndexs[sonL] = allIndexs[root];
                    allIndexs[root] = sonR;
                    root = sonL;
                } else {
                    break;
                }
            } else {
                if (allIndexs[root] < allIndexs[sonR]) {
                    sonL = allIndexs[sonR];
                    allIndexs[sonR] = allIndexs[root];
                    allIndexs[root] = sonL;
                    root = sonR;
                } else {
                    break;
                }
            }
        }
    }
}
__global__ void kernelRenderPixel(int *allIndexs) {
    int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelY = blockIdx.y * blockDim.y + threadIdx.y;
    int imageWidth = cuConstRendererParams.imageWidth;
    int imageHeight = cuConstRendererParams.imageHeight;
    if (pixelX >= imageWidth || pixelY >= imageHeight)
        return;
    int pieceX = pixelX / cuConstRendererParams.widthPerPiece;
    int pieceY = pixelY / cuConstRendererParams.heightPerPiece;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    int pieceIndex = pieceX + pieceY * cuConstRendererParams.numPieceWidth;
    int numCircle = cuConstRendererParams.numElementInPieceCurrent[pieceIndex];
    int priorNum = cuConstRendererParams.prefixSum[pieceIndex];
    allIndexs += priorNum;

    for (int i = 0; i < numCircle; i++) {
        int index = allIndexs[i];
        int index3 = 3 * index;
        // read position and radius
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        // float  rad = cuConstRendererParams.radius[index];
        // 像素信息
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + pixelX)]);
        float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
        // 更新像素信息
        shadePixel(index, pixelCenterNorm, p, imgPtr);
    }
}

void
CudaRenderer::render() {
    hipError_t cudaStatus;
    double startTime, endTime;
    int numPiece = numPieceWidth * numPieceHeight;
    // 将整个图片分块进行(块数不能太小，否则需要更改kernelRenderPixel函数相关的调用)
    printf("---------------------------------------------\n");
    printf("CudaRenderer::render start\n");
    printf("numCircles: %d \n", numCircles);
    printf("图片将分成 %d * %d 块\n", numPieceWidth, numPieceHeight);
    printf("每块包含 %d * %d 个像素点\n", widthPerPiece, heightPerPiece);

    // 圆并行
    printf("---------------------------------------------\n");
    printf("求每个块与之对应的圆的数量(numCircleInPiece)\n");
    dim3 blockDimCircle(256, 1);
    dim3 gridDimCircle((numCircles + blockDimCircle.x - 1) / blockDimCircle.x);
    printf("gridDimCircle: %d, blockDimCircle: %d\n", gridDimCircle.x, blockDimCircle.x);
    startTime = CycleTimer::currentSeconds();
    numCircleInPiece<<<gridDimCircle, blockDimCircle>>>();
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("numCirclePiece final, run time: %.3f ms\n", 1000.f * (endTime - startTime));

    // 根据结果开辟空间
    printf("---------------------------------------------\n");
    printf("申请新空间(mallocNewMemory)\n");
    startTime = CycleTimer::currentSeconds();
    int newMemorySize = exclusive_scan(cudaDeviceElementInPiece, numPiece, cudaDevicePrefixSum);
    printf("newMemorySize: %d\n", newMemorySize);
    int* allIndexs;
    cudaStatus = hipMalloc((void**)&allIndexs, sizeof(int) * newMemorySize);
    if (cudaStatus != hipSuccess) 
        printf("mallocNewMemory hipMalloc error: %d\n", cudaStatus);

    endTime = CycleTimer::currentSeconds();
    printf("mallocNewMemory final, run time: %.3f ms\n", 1000.f * (endTime - startTime));

    // 二次圆并行
    printf("---------------------------------------------\n");
    printf("将圆的ID添加至新申请的空间(addIndexToNewMemory)\n");
    printf("gridDimCircle: %d, blockDimCircle: %d\n", gridDimCircle.x, blockDimCircle.x);
    startTime = CycleTimer::currentSeconds();
    addIndexToNewMemory<<<gridDimCircle, blockDimCircle>>>(allIndexs);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("addIndexToNewMemory final, run time: %.3f ms\n", 1000.f * (endTime - startTime));

    // 块并行排序
    printf("---------------------------------------------\n");
    printf("每个块对圆id排序(sortPerPiece)\n");
    dim3 blockDimPiece(std::min(256, numPiece), 1);
    dim3 gridDimPiece((numPiece + blockDimPiece.x - 1) / blockDimPiece.x);
    printf("gridDimPiece: %d, blockDimPiece: %d\n", gridDimPiece.x, blockDimPiece.x);
    startTime = CycleTimer::currentSeconds();
    sortPerPiece<<<blockDimPiece, gridDimPiece>>>(allIndexs);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("sortPerPiece final, run time: %.3f ms\n", 1000.f * (endTime - startTime));

    // 像素并行计算
    printf("---------------------------------------------\n");
    printf("开始像素并行计算\n");
    dim3 blockDimPixel(std::min(widthPerPiece, 8), std::min(heightPerPiece, 8));
    dim3 gridDimPixel((image->width + blockDimPixel.x - 1) / blockDimPixel.x, 
                      (image->height + blockDimPixel.y - 1) / blockDimPixel.y);
    printf("gridDimPixel.x: %d, blockDimPixel.x: %d\n", gridDimPixel.x, blockDimPixel.x);
    printf("gridDimPixel.y: %d, blockDimPixel.y: %d\n", gridDimPixel.y, blockDimPixel.y);
    startTime = CycleTimer::currentSeconds();
    kernelRenderPixel<<<gridDimPixel, blockDimPixel>>>(allIndexs);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("kernelRenderPixel final, run time: %.3f ms\n", 1000.f * (endTime - startTime));
    printf("---------------------------------------------\n");
    // 记得释放空间
    hipFree(allIndexs);
}

