#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"
#include "CycleTimer.h"

#define NUM_PIECE_WIDTH 32
#define NUM_PIECE_HEIGHT 32

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;

    // 分块
    int* numElementInPiece;
    int* numElementInPieceCurrent;
    int** newMemoryToPiece;
    int numPieceWidth;
    int numPieceHeight;
    int widthPerPiece;
    int heightPerPiece;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(index, pixelCenterNorm, p, imgPtr);
            imgPtr++;
        }
    }
}

__device__ __inline__ int
circleInBoxConservative(
    float circleX, float circleY, float circleRadius,
    float boxL, float boxR, float boxT, float boxB)
{

    // expand box by circle radius.  Test if circle center is in the
    // expanded box.

    if ( circleX >= (boxL - circleRadius) &&
         circleX <= (boxR + circleRadius) &&
         circleY >= (boxB - circleRadius) &&
         circleY <= (boxT + circleRadius) ) {
        return 1;
    } else {
        return 0;
    }
}

__device__ __inline__ int
circleInBox(
    float circleX, float circleY, float circleRadius,
    float boxL, float boxR, float boxT, float boxB)
{

    // clamp circle center to box (finds the closest point on the box)
    float closestX = (circleX > boxL) ? ((circleX < boxR) ? circleX : boxR) : boxL;
    float closestY = (circleY > boxB) ? ((circleY < boxT) ? circleY : boxT) : boxB;

    // is circle radius less than the distance to the closest point on
    // the box?
    float distX = closestX - circleX;
    float distY = closestY - circleY;

    if ( ((distX*distX) + (distY*distY)) <= (circleRadius*circleRadius) ) {
        return 1;
    } else {
        return 0;
    }
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
    cudaDeviceElementInPiece = NULL;
    cudaDeviceNewMemoryToPiece = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
        hipFree(cudaDeviceElementInPiece);
        hipFree(cudaDeviceElementInPieceCurrent);
        hipFree(cudaDeviceNewMemoryToPiece);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    // 具体分块信息
    numPieceWidth = NUM_PIECE_WIDTH;
    numPieceHeight = NUM_PIECE_HEIGHT;
    widthPerPiece = (image->width + numPieceWidth - 1) / numPieceWidth;
    heightPerPiece = (image->height + numPieceHeight - 1) / numPieceHeight;
    printf("widthPerPiece: %d, heightPerPiece: %d\n", widthPerPiece, heightPerPiece);
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&cudaDeviceElementInPiece, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");
    cudaStatus = hipMalloc((void**)&cudaDeviceElementInPieceCurrent, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");
    cudaStatus = hipMalloc((void**)&cudaDeviceNewMemoryToPiece, sizeof(int*) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMalloc failed\n");
    hipMemset(cudaDeviceElementInPiece, 0, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMemset failed\n");
    hipMemset(cudaDeviceElementInPieceCurrent, 0, sizeof(int) * numPieceHeight * numPieceWidth);
    if (cudaStatus != hipSuccess) 
        fprintf(stderr, "hipMemset failed\n");

    hipDeviceSynchronize();

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    params.numPieceWidth = numPieceWidth;
    params.numPieceHeight = numPieceHeight;
    params.widthPerPiece = widthPerPiece;
    params.heightPerPiece = heightPerPiece;
    params.numElementInPiece = cudaDeviceElementInPiece;
    params.numElementInPieceCurrent = cudaDeviceElementInPieceCurrent;
    params.newMemoryToPiece = cudaDeviceNewMemoryToPiece;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();                                                                     
}


__global__
void numCircleInPiece() {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= cuConstRendererParams.numCircles) {
        return;
    }
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short numPieceWidth = cuConstRendererParams.numPieceWidth;
    short numPieceHeight = cuConstRendererParams.numPieceHeight;
    short widthEachPiece = (imageWidth + numPieceWidth - 1) / numPieceWidth;
    short heightEachPiece = (imageHeight + numPieceHeight - 1) / numPieceHeight;
    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index * 3]);
    float  rad = cuConstRendererParams.radius[index];

    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    short pieceMinX = screenMinX / widthEachPiece;
    short pieceMaxX = (screenMaxX - 1) / widthEachPiece + 1;
    short pieceMinY = screenMinY / heightEachPiece;
    short pieceMaxY = (screenMaxY - 1) / heightEachPiece + 1;

    float invWidth = 1.f / imageWidth * widthEachPiece;
    float invHeight = 1.f / imageHeight * heightEachPiece;

    // 遍历像素块
    float boxB = invHeight * pieceMinY;
    for (int pieceY = pieceMinY; pieceY < pieceMaxY; pieceY++, boxB += invHeight) {
        int pieceIndex = pieceY * numPieceWidth + pieceMinX;
        int* val = &cuConstRendererParams.numElementInPiece[pieceIndex];
        float boxL = invWidth * pieceMinX;
        for (int pieceX = pieceMinX; pieceX < pieceMaxX; pieceX++, val++, boxL += invWidth) {
            if (circleInBox(p.x, p.y, rad, boxL, boxL + invWidth, boxB + invHeight, boxB)) {
                atomicAdd(val, 1);
            }
        }
    }
}
// void CudaRenderer::numCircleInPieceCPU(std::vector<int>&cmp_v1, std::vector<std::vector<int>>& cmp_v2) {
//     int imageWidth = image->width;
//     int imageHeight = image->height;
//     int widthEachPiece = (imageWidth + numPieceWidth - 1) / numPieceWidth;
//     int heightEachPiece = (imageHeight + numPieceHeight - 1) / numPieceHeight;
//     // printf("%d, %d, %d, %d, %d, %d\n", imageWidth, imageHeight, widthEachPiece, heightEachPiece, numPieceWidth, numPieceHeight);
//     for (int index = 0;index < this->numCircles;index++) {
//         // read position and radius
//         float3 p = *(float3*)(&position[index * 3]);
//         float  rad = radius[index];

//         int minX = static_cast<short>(imageWidth * (p.x - rad));
//         int maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
//         int minY = static_cast<short>(imageHeight * (p.y - rad));
//         int maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

//         // a bunch of clamps.  Is there a CUDA built-in for this?
//         int screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
//         int screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
//         int screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
//         int screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

//         int pieceMinX = screenMinX / widthEachPiece;
//         int pieceMaxX = (screenMaxX - 1) / widthEachPiece + 1;
//         int pieceMinY = screenMinY / heightEachPiece;
//         int pieceMaxY = (screenMaxY - 1) / heightEachPiece + 1;
//         // printf("%d, %d, %d, %d\n", screenMinX, screenMaxX, screenMinY, screenMaxY);
//         // printf("%d, %d, %d, %d\n", pieceMinX, pieceMaxX, pieceMinY, pieceMaxY);

//         // 遍历像素块
//         for (int pieceY = pieceMinY; pieceY < pieceMaxY; pieceY++) {
//             int pieceIndex = pieceY * numPieceWidth + pieceMinX;
//             // printf("%d\n", pieceIndex);
//             for (int pieceX = pieceMinX; pieceX < pieceMaxX; pieceX++, pieceIndex++) {
//                 // 修改部分
//                 // cmp_v1[pieceIndex]++;
//                 // cmp_v2[pieceIndex].push_back(index);
//             }
//         }
//     }
// }

__global__
void addIndexToNewMemory() {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= cuConstRendererParams.numCircles) {
        return;
    }
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short numPieceWidth = cuConstRendererParams.numPieceWidth;
    short numPieceHeight = cuConstRendererParams.numPieceHeight;
    short widthEachPiece = (imageWidth + numPieceWidth - 1) / numPieceWidth;
    short heightEachPiece = (imageHeight + numPieceHeight - 1) / numPieceHeight;
    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index * 3]);
    float  rad = cuConstRendererParams.radius[index];

    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    short pieceMinX = screenMinX / widthEachPiece;
    short pieceMaxX = (screenMaxX - 1) / widthEachPiece + 1;
    short pieceMinY = screenMinY / heightEachPiece;
    short pieceMaxY = (screenMaxY - 1) / heightEachPiece + 1;

    float invWidth = 1.f / imageWidth * widthEachPiece;
    float invHeight = 1.f / imageHeight * heightEachPiece;
    // 新的地址空间合集
    int** newMemoryToPiece = cuConstRendererParams.newMemoryToPiece;

    float boxB = invHeight * pieceMinY;
    // 遍历像素块
    for (int pieceY = pieceMinY; pieceY < pieceMaxY; pieceY++, boxB += invHeight) {
        int pieceIndex = pieceY * numPieceWidth + pieceMinX;
        float boxL = invWidth * pieceMinX;
        for (int pieceX = pieceMinX; pieceX < pieceMaxX; pieceX++, pieceIndex++, boxL += invWidth) {
            if (circleInBox(p.x, p.y, rad, boxL, boxL + invWidth, boxB + invHeight, boxB)) {
                int* val = &cuConstRendererParams.numElementInPieceCurrent[pieceIndex];
                int* newMemory = newMemoryToPiece[pieceIndex];
                // 修改部分
                int d = atomicAdd(val, 1);
                newMemory[d] = index;
            }
        }
    }
}
__global__
void sortPerPiece() {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int numPieceWidth = cuConstRendererParams.numPieceWidth;
    int numPieceHeight = cuConstRendererParams.numPieceHeight;
    if (index >= numPieceWidth * numPieceHeight) {
        return;
    }
    // 获取地址
    int total = cuConstRendererParams.numElementInPiece[index];
    int *arr = cuConstRendererParams.newMemoryToPiece[index];
    // 特判，会出现一个地方的空间为 0 的情况;
    if (total <= 0) {
        return;
    }
    // 堆排序
    // 变大根堆
    for (int i = total - 1;i >= 0;i--) {
        int root = i;
        while(true) {
            int sonL = (root << 1) + 1;
            int sonR = sonL + 1;
            if (sonL >= total) {
                break;
            }
            if (sonR >= total || arr[sonL] >= arr[sonR]) {
                if (arr[root] < arr[sonL]) {
                    sonR = arr[sonL];
                    arr[sonL] = arr[root];
                    arr[root] = sonR;
                    root = sonL;
                } else {
                    break;
                }
            } else {
                if (arr[root] < arr[sonR]) {
                    sonL = arr[sonR];
                    arr[sonR] = arr[root];
                    arr[root] = sonL;
                    root = sonR;
                } else {
                    break;
                }
            }
        }
    }

    while(--total) {
        int root = arr[0];
        arr[0] = arr[total];
        arr[total] = root;
        root = 0;
        while(true) {
            int sonL = (root << 1) + 1;
            int sonR = sonL + 1;
            if (sonL >= total) {
                break;
            }
            if (sonR >= total || arr[sonL] >= arr[sonR]) {
                if (arr[root] < arr[sonL]) {
                    sonR = arr[sonL];
                    arr[sonL] = arr[root];
                    arr[root] = sonR;
                    root = sonL;
                } else {
                    break;
                }
            } else {
                if (arr[root] < arr[sonR]) {
                    sonL = arr[sonR];
                    arr[sonR] = arr[root];
                    arr[root] = sonL;
                    root = sonR;
                } else {
                    break;
                }
            }
        }
    }
    // for (int i = 1;i < cuConstRendererParams.numElementInPiece[index];i++) {
    //     if (arr[i - 1] >= arr[i]) {
    //         printf("arr[%d - 1] >= arr[%d], arr[i - 1] = %d, arr[i] = %d\n", i, i, arr[i - 1], arr[i]);
    //     }
    //     assert(arr[i - 1] < arr[i]);
    // }
}
__global__ void kernelRenderPixel() {
    int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelY = blockIdx.y * blockDim.y + threadIdx.y;
    int imageWidth = cuConstRendererParams.imageWidth;
    int imageHeight = cuConstRendererParams.imageHeight;
    if (pixelX >= imageWidth || pixelY >= imageHeight)
        return;
    int pieceX = pixelX / cuConstRendererParams.widthPerPiece;
    int pieceY = pixelY / cuConstRendererParams.heightPerPiece;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    int pieceIndex = pieceX + pieceY * cuConstRendererParams.numPieceWidth;
    assert(pieceIndex < cuConstRendererParams.numPieceWidth * cuConstRendererParams.numPieceHeight);
    int numCircle = cuConstRendererParams.numElementInPiece[pieceIndex];
    int *circleIndexs = cuConstRendererParams.newMemoryToPiece[pieceIndex];

    for (int i = 0; i < numCircle; i++) {
        int index = circleIndexs[i];
        int index3 = 3 * index;
        // read position and radius
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        // float  rad = cuConstRendererParams.radius[index];
        // 像素信息
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + pixelX)]);
        float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
        // 更新像素信息
        shadePixel(index, pixelCenterNorm, p, imgPtr);
    }
}

void
CudaRenderer::render() {
    hipError_t cudaStatus;
    double startTime, endTime;
    // 将整个图片分块进行(块数不能太小，否则需要更改kernelRenderPixel函数相关的调用)
    printf("---------------------------------------------\n");
    printf("CudaRenderer::render start\n");
    printf("图片将分成 %d * %d 块\n", numPieceWidth, numPieceHeight);
    printf("每块包含 %d * %d 个像素点\n", widthPerPiece, heightPerPiece);

    dim3 blockDimCircle(256, 1);
    dim3 gridDimCircle((numCircles + blockDimCircle.x - 1) / blockDimCircle.x);
    // 圆并行
    printf("---------------------------------------------\n");
    printf("求每个块与之对应的圆的数量(numCircleInPiece)\n");
    // std::vector<int>cmp_v1(1024, 0);
    // std::vector<std::vector<int>> cmp_v2(1024, std::vector<int>());
    // numCircleInPieceCPU(cmp_v1, cmp_v2);
    // printf("numCirclePieceCPU final\n");
    printf("gridDimCircle: %d, blockDimCircle: %d\n", gridDimCircle.x, blockDimCircle.x);
    startTime = CycleTimer::currentSeconds();
    numCircleInPiece<<<gridDimCircle, blockDimCircle>>>();
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("numCirclePiece final, run time: %.3f ms\n", 1000.f * (endTime - startTime));
    int numPiece = numPieceWidth * numPieceHeight;
    int* elementInPiece = new int[numPiece];
    hipMemcpy(elementInPiece, cudaDeviceElementInPiece, sizeof(int) * numPiece, hipMemcpyDeviceToHost);
    // for (int i = 0;i < numPiece;i++) {
    //     printf("第 %d 块: 有 %d 个圆\n", i, elementInPiece[i]);
    // }
    // for(int i = 0;i < numPiece;i++) {
    //     if(elementInPiece[i] != cmp_v1[i]) {
    //         printf("pieceIndex: %d, expect: %d, value: %d\n", i, cmp_v1[i], elementInPiece[i]);
    //         return;
    //     }
    // }
    // printf("result is correct\n\n");

    // 根据结果开辟空间
    printf("---------------------------------------------\n");
    printf("申请新空间(mallocNewMemory, 串行)\n");
    startTime = CycleTimer::currentSeconds();
    int **newMemory = new int*[numPiece];
    for (int i = 0;i < numPieceWidth;i++) {
        for (int j = 0;j < numPieceHeight;j++) {
            int index = i + j * numPieceWidth;
            cudaStatus = hipMalloc((void**)&newMemory[index], sizeof(int) * elementInPiece[index]);
            // printf("第 %d 块: 新内存开辟完成\n", index);
            if (cudaStatus != hipSuccess) 
                printf("mallocNewMemory hipMalloc error: %d\n", cudaStatus);
        }
    }
    cudaStatus = hipMemcpy(cudaDeviceNewMemoryToPiece, newMemory, sizeof(int*) * numPiece, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
        printf("mallocNewMemory hipMalloc error: %d\n", cudaStatus);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("mallocNewMemory final, run time: %.3f ms\n", 1000.f * (endTime - startTime));

    // 二次圆并行
    printf("---------------------------------------------\n");
    printf("将圆的ID添加至新申请的空间(addIndexToNewMemory)\n");
    printf("gridDimCircle: %d, blockDimCircle: %d\n", gridDimCircle.x, blockDimCircle.x);
    startTime = CycleTimer::currentSeconds();
    addIndexToNewMemory<<<gridDimCircle, blockDimCircle>>>();
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("addIndexToNewMemory final, run time: %.3f ms\n", 1000.f * (endTime - startTime));
    // 这里需要写一点验证函数
    // for (int i = 0;i < numPiece;i++) {
    //     int* arr = new int[elementInPiece[i]];
    //     hipMemcpy(arr, newMemory[i], sizeof(int) * elementInPiece[i], hipMemcpyDeviceToHost);
    //     printf("第 %d 块的所有圆 id: ", i);
    //     for (int j = 0;j < elementInPiece[i];j++) {
    //         printf("%d ", arr[j]);
    //     }
    //     printf("\n");
    //     delete[] arr;
    // }
    // printf("result is correct\n\n");

    // 块并行排序
    printf("---------------------------------------------\n");
    printf("每个块对圆id排序(sortPerPiece)\n");
    dim3 blockDimPiece(std::min(256, numPiece), 1);
    dim3 gridDimPiece((numPiece + blockDimPiece.x - 1) / blockDimPiece.x);
    printf("gridDimPiece: %d, blockDimPiece: %d\n", gridDimPiece.x, blockDimPiece.x);
    startTime = CycleTimer::currentSeconds();
    sortPerPiece<<<blockDimPiece, gridDimPiece>>>();
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("sortPerPiece final, run time: %.3f ms\n", 1000.f * (endTime - startTime));
    // 验证部分
    // for (int i = 0;i < numPiece;i++) {
    //     int* arr = new int[elementInPiece[i]];
    //     hipMemcpy(arr, newMemory[i], sizeof(int) * elementInPiece[i], hipMemcpyDeviceToHost);
    //     printf("排序后第 %d 块的所有圆 id: ", i);
    //     for (int j = 0;j < elementInPiece[i];j++) {
    //         printf("%d ", arr[j]);
    //     }
    //     printf("\n");
    //     delete[] arr;
    // }
    // 像素并行计算
    printf("---------------------------------------------\n");
    printf("开始像素并行计算\n");
    dim3 blockDimPixel(std::min(widthPerPiece, 8), std::min(heightPerPiece, 8));
    dim3 gridDimPixel((image->width + blockDimPixel.x - 1) / blockDimPixel.x, 
                      (image->height + blockDimPixel.y - 1) / blockDimPixel.y);
    printf("gridDimPixel.x: %d, blockDimPixel.x: %d\n", gridDimPixel.x, blockDimPixel.x);
    printf("gridDimPixel.y: %d, blockDimPixel.y: %d\n", gridDimPixel.y, blockDimPixel.y);
    startTime = CycleTimer::currentSeconds();
    kernelRenderPixel<<<gridDimPixel, blockDimPixel>>>();
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("kernelRenderPixel final, run time: %.3f ms\n", 1000.f * (endTime - startTime));
    printf("---------------------------------------------\n");
    // 记得释放空间
    delete[] elementInPiece;
    delete[] newMemory;
}

